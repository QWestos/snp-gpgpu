#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <stdint.h>
#include <stdlib.h>

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
	int width;
	int height;
	float* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 8

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);


// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// Load A and B to device memory
	Matrix d_A;
	d_A.width = A.width; d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	Matrix d_B;
	d_B.width = B.width; d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
// Allocate C in device memory
	Matrix d_C;
	d_C.width = C.width; d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc(&d_C.elements, size);
// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
// Read C from device memory
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
// Free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
// Each thread computes one element of C
// by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int e = 0; e < A.width; ++e)
		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
	C.elements[row * C.width + col] = Cvalue;
}

int main (void){


	Matrix A;
	A.width = BLOCK_SIZE;
	A.height = BLOCK_SIZE;
	float elements[BLOCK_SIZE * BLOCK_SIZE] = {  1.0, 0.0, 1.0, 1.0, 0.0, 0.0, 0.0, 0.0,  0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,  0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,  0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 };
	A.elements = elements;
	

	Matrix B;
	B.width = BLOCK_SIZE;
	B.height = BLOCK_SIZE;
	float Belements[BLOCK_SIZE * BLOCK_SIZE] = {  -1, 1, 1, 0, 0, 0, 0, 0,  -2, 1, 1, 0, 0, 0, 0, 0,  1, -1, 1, 0, 0, 0, 0, 0,  0, 0, -1, 0, 0, 0, 0, 0,  0, 0, -2, 0, 0, 0, 0, 0,  0, 0, 0, 0, 0, 0, 0, 0,  0, 0, 0, 0, 0, 0, 0, 0,  0, 0, 0, 0, 0, 0, 0, 0 };
	B.elements = Belements;
	//int B[BLOCK_SIZE][BLOCK_SIZE] = { { -1, 1, 1, 0, 0, 0, 0, 0}, { -2, 1, 1, 0, 0, 0, 0, 0}, { 1, -1, 1, 0, 0, 0, 0, 0}, { 0, 0, -1, 0, 0, 0, 0, 0}, { 0, 0, -2, 0, 0, 0, 0, 0}, { 0, 0, 0, 0, 0, 0, 0, 0}, { 0, 0, 0, 0, 0, 0, 0, 0}, { 0, 0, 0, 0, 0, 0, 0, 0} }; 
	
	Matrix C;
	C.width = BLOCK_SIZE;
	C.height = BLOCK_SIZE;
	//float Belements[BLOCK_SIZE * BLOCK_SIZE] = {  -1, 1, 1, 0, 0, 0, 0, 0,  -2, 1, 1, 0, 0, 0, 0, 0,  1, -1, 1, 0, 0, 0, 0, 0,  0, 0, -1, 0, 0, 0, 0, 0,  0, 0, -2, 0, 0, 0, 0, 0,  0, 0, 0, 0, 0, 0, 0, 0,  0, 0, 0, 0, 0, 0, 0, 0,  0, 0, 0, 0, 0, 0, 0, 0 };
	//C.elements = Belements;

	MatMul( A, B, C);

	//print array C
	int l  = BLOCK_SIZE * BLOCK_SIZE;
	for(  int i = 0; i < l; i++) {
		printf( "\n");
		printf ( " %f  ", C.elements[i] );
		printf( "\n");
	}
}
