
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctype.h>

int main ( int argc, char *argv[ ] ){
	if ( argc != 2 )
		printf( "\n Usage \n%s filetoread \n", argv[ 0 ] );
	else {
		char x; 
		int rules[ 100 ]; //length of rules that can be read
		int a = 0;
		char *filename = argv[ 1 ];
		FILE *ptr1 = fopen( filename, "r" );
		fscanf( ptr1, "%c", &x  );
		while( !feof( ptr1 ) ) {
			if ( isalnum( x ) ) {
				int y = atoi( &x );
				//printf( "\n%d", y );
				fscanf( ptr1, "%c", &x );
				rules[ a ] = y;
			}
			else { // ! = 33, $ = 36, ' ' = 32
				//printf( "-ELSE-" );
				rules[ a ] = -1;
				fscanf( ptr1, "%c", &x );
			}
			a++;
		}
	//print the loaded rules
	// Rules on file: 2 2 $ 1 $ 1 2
	// Rules on load: |2 |-1 |2 |-1 |-1 |-1 |1 |-1 |-1 |-1 |1 |-1 |2 |-1 |
		printf( "\n" );
		int oneCnt = 1;
		int ruleCnt = 1;
		int neuron = 1;
		// Find out how many rules are there.
		for( int x = 0; rules[ x ] != 0; x++) {
			//printf( "%d |", rules[ x ] );
			if ( rules[ x ] > 0 && oneCnt < 4 ){
				oneCnt = 1;
			}
			else if ( rules[ x ] < 0 && oneCnt < 3 ) {
				oneCnt = oneCnt + 1;
			}
			else if ( rules[ x ] < 0 && oneCnt == 3 ) {
				oneCnt = 1;
				ruleCnt = ruleCnt + 1;
			}
		}
		oneCnt = 1;
		printf( "\nThere are %d rules loaded\n", ruleCnt );
		int rulePrint = 1;
		printf( "Neuron %d rule/s:\n", neuron );
		for( int x = 0; rulePrint <= ruleCnt && rules[ x ] != 0; x++) {
			//printf( "%d |", rules[ x ] );
			if ( rules[ x ] > 0 && oneCnt < 4 ){
				printf( " %d ", rules[ x ], oneCnt, rulePrint );
				oneCnt = 1;
			}
			else if ( rules[ x ] < 0 && oneCnt < 3 ) {
				oneCnt = oneCnt + 1;
//				printf( " B " );
			}
			else if ( rules[ x ] < 0 && oneCnt == 3 ) {
				neuron = neuron + 1;
				printf( "\nNeuron %d rule/s:\n", neuron );
				oneCnt = 1;
				rulePrint = rulePrint + 1;
//				printf( " C " );
			}
		} 
	//	printf( " '%c' '%d' '%c' '%d' '%c' '%d'", " ", " ", "$", "$", 33, 33 );
		printf( "\n" );
	}	
}
