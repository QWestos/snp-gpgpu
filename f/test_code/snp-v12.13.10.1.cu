
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <cuda.h>
//#include "cuda_runtime_api.h"
//#include <stdint.h>
//#include <stdlib.h>

//This is the working matrix multiplication code - very basic
/*
Done:
- printing of matrix in a more pleasant manner using printMatrix function
- command line arguments
- opens matrix files and reads the matrix successfully
- working array passing from main to auxiliary (loadMatrixFile) function :)
- fixed printing of matrix
- fixed erroneous matrix values by moving loading into host matrix multiplication function!
- basic move towards SN P simulation: multiplication of s0 and Msnp
- moving from multiplication to finally simulating an SNP (sort of)

Problems:
- (fixed)  MatA and MatB values are overlapping and erroneous
*/


// START of AUXILIARY functions

//START vector addition kernel function
__global__ void MatrixAddKernel ( int  *Md, int *Nd, int *Pd, int N ){
        int tid = blockIdx.x; //thred id
	if ( tid < N )
		Pd[ tid ] = Md[ tid ] + Nd[ tid ];
/*        int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ k * Width + tx ];
		Pvalue = Mdelement + Ndelement;
	}
        Pd[ ty * Width + tx  ] = Pvalue; */
}							
//END of kernel addition


//Start of kernel multiplication
__global__ void MatrixMulKernel ( int  *Md, int *Nd, int *Pd, int Width ){
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ k * Width + tx ];
		Pvalue += Mdelement * Ndelement;
	}

	Pd[ ty * Width + tx  ] = Pvalue;
}
//End of kernel multiplication


//function to print matrix
void printMatrix ( int *M, int rows, int columns ){
	//assumes matrix is in row-major format
	int index;
	printf ( "\n \n " );
	for ( int v = 0; v < rows; v++  ){
	//assumes a square matrix
		for ( int w = 0; w < columns; w++   ) {
			index = v * columns + w;
			printf ( " %02d", M[ index ]  );
		}
		printf ( " \n\n " );
	}
}//End of printMatrix function


//START of loadMatrixFile
void loadMatrixFile( char *filename, int *z, int matWidth, int matHeight ){
	int y = 0;
	int w = 0;
	int x;
	int offset = 0;
	FILE *ptr1 = fopen( filename, "r" );
//	int *z = ( int * )malloc( sizeof( ( matWidth * matHeight ) ) );
	//int z[ ( matWidth * matHeight ) + offset ] ;
	fscanf( ptr1, " %d", &x  );
	while( !feof( ptr1 ) && y < ( matWidth * matHeight ) + 1 ){
		if ( y > offset ){
			fscanf( ptr1, " %d", &z[ w - offset ]  );
			//printf( " B: z[ %d ]: %d \n", w, z[ w - offset ] );
			w++;
		}
		else{
			fscanf( ptr1, " %d", &x );
		}
		y++;
	}
	fclose( ptr1 );
//	x = y = w = 0;
//	array = &z[ 0 ];
//	free( z );
}
//END of loadMatrixFile


//Start of matrix multiplication host function MatrixMul
void MatrixMul( char *filename0, char *filename1, char *filename2, int Width /*, int *M, int *N, int  *P, int Width*/ ){
	int size = Width * Width * sizeof( int );
	int *Md, *Nd, *Pd;

	dim3 dimBlock( Width, Width );
	dim3 dimGrid( 1, 1 );

	int *matA = ( int * )malloc( size );
	//printf( "Width and height of Matrix A: %d %d and init values are\n", Width, Width );
	//printMatrix( matA, Width, Width );
	loadMatrixFile( filename1, matA, Width, Width );

	printf( " \ns after loading from file: \n" );
	printMatrix( matA, Width, Width );
			
	int *matB = ( int * )malloc( size );
	loadMatrixFile( filename2, matB, Width, Width );
		
	printf( " \nM after loading from file: \n" );
	printMatrix( matB, Width, Width );
			
	//assumes a square matrix
	int *matC = ( int * )malloc( size );
	
	hipMalloc( (void**) &Md, size );
	hipMemcpy( Md, matA, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Nd, size );
        hipMemcpy( Nd, matB, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Pd, size );	
	
	MatrixMulKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );
	//MatrixAddKernel<<< N, 1 >>>( Md, Nd, Pd );

	hipMemcpy( matC, Pd, size, hipMemcpyDeviceToHost );

	printf( " \ns * M: \n" );
	printMatrix( matC, Width, Width );

	free( matA ); free( matB ); free( matC );
	hipFree( Md ); hipFree( Nd ); hipFree ( Pd );
}
//End of Matrix multiplication function MatrixMul


//END of AUXILIARY functions


//START of MAIN function
int main ( void ) {
	int x;
	while( x != 3) {
		printf( "\n Type \n 1 to enter filenames < 20 in length \n 2 for 2 \n 3 to quit \n: " );
		scanf( "%d", &x );
		switch( x ){
			case 1:
				char a[ 20 ], b[ 20 ], c[ 20 ];
				int d;
				printf( " Please enter spiking vector file: \n" );
				scanf( " %s", &a );
				printf( " Please enter configuration vector file: \n" );
				scanf( " %s", &b );
				printf( " Please enter spiking transition matrix file: \n" );
				scanf( " %s", &c ); 
				printf( " Please enter the square matrix width: \n" );
				scanf( " %d", &d ); 

				if( ( strlen( a ) ) > 20 && ( strlen( b ) ) > 20 && ( strlen( c ) ) > 20  ) {
					printf( " Filename/s was/were too long ( > 20 char )  " );
					// Do something about segmentation fault here
					//spikVec = { "\0" }; // doesn't work
					//*confVec = NULL; // doesn't work
					break;
				}
				else {
					printf( " You entered the file %s for the spiking vector \n", a );
					printf( " You entered the file %s for the configuration vector \n", b );
					printf( " You entered the file %s for the spiking transition matrix \n ", c );
					char *confVec = b;
					char *spikVec = a;
					char *spikTransMat = c;
					int width = d;
		
					printf( "\nYou have entered files %s, %s, and %s and square matrix width %d \n", spikVec, confVec, spikTransMat, width );

					//load matrices from files
					FILE *ptr1 = fopen( confVec, "r" );
					FILE *ptr2 = fopen( spikVec, "r" );
					FILE *ptr3 = fopen( spikTransMat, "r" );

					if ( ptr1 == 0 && ptr2 == 0 && ptr3 == 0 )
						printf( "\n could not open one of the following files: %s %s %s \n", a, b, c );
					else {
						MatrixMul( confVec, spikVec, spikTransMat, width );
					}
					fclose( ptr1 ); fclose( ptr2 ); fclose( ptr3 );
					break;				
				}
			case 2: 
				printf( " You entered 2 \n" );
				break;
			case 3: 
				printf( " You entered quit. Bye! \n" );
				break;
			default:
				printf( " You entered an invalid choice \n" );
				break;
		}

	}
}
//END of MAIN function
