
#include <hip/hip_runtime.h>
#include <stdio.h>
 
//#include <cuda.h>
//#include "cuda_runtime_api.h"
//#include <stdint.h>
//#include <stdlib.h>

//This is the working matrix multiplication code - very basic
/****

Done:
- printing of matrix in a more pleasant manner using printMatrix function
- command line arguments
- opens matrix files and reads the matrix successfully
- working array passing from main to auxiliary (loadMatrixFile) function :)
- fixed printing of matrix
- fixed erroneous matrix values by moving loading into host matrix multiplication function!
- basic move towards SN P simulation: multiplication of s0 and Msnp
- moving from multiplication to finally simulating an SN P (sort of) in a very basic manner
- MatrixAddKernel now works :)
- Can now do Ck+1 = Ck + sk * M :)
- outputs Ck+1 to a file whose filename is entered by the user

Problems:
- (fixed)  MatA and MatB values are overlapping and erroneous

TODOS:
- write Ck+1 to an output file ( done )
- error checking of switch case input ( scanf of int and char )
- use multiple files + make file
- see code comments

****/


/***
**** START of AUXILIARY functions
***/

/*
START of KERNEL functions
*/
//START vector addition kernel function
void MatrixAddKernel ( int  *Md, int *Nd, int *Pd, int Width ){
	
	//due to row-major ordering of matrix elements
	//int Pvalue = 0;
	for ( int ty = 0; ty < Width; ++ty ){
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ ty * Width + k ];
		Pd[ ty * Width + k ] = Mdelement + Ndelement;
	}
	}
	//Pd[ ty * Width + tx  ] = Pvalue;	
}

/*
__global__ void MatrixAddKernel ( int  *Md, int *Nd, int *Pd, int Width ){
	// MatrixAddKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );
	//dim3 dimBlock( Width, Width ); dim3 dimGrid( 1, 1 );
	//int tx = threadIdx.x;
	int ty = threadIdx.y;
	//due to row-major ordering of matrix elements
	//int Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ ty * Width + k ];
		Pd[ ty * Width + k ] = Mdelement + Ndelement;
	}
	//Pd[ ty * Width + tx  ] = Pvalue;
} */							
//END of kernel addition


//START of kernel multiplication
void MatrixMultiplication( int *Md, int *Nd, int *Pd, int Width)
{
	for (int i = 0; i < Width; ++i)
		for (int j = 0; j < Width; ++j) {
			int sum = 0;
			for (int k = 0; k < Width; ++k) {
				int a = Md[i * Width + k];
				int b = Nd[k * Width + j];
				sum += a * b;
			}
			Pd[i * Width + j] = sum;
		}
}
/*__global__ void MatrixMulKernel ( int  *Md, int *Nd, int *Pd, int Width ){
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	//due to row-major ordering of matrix elements
	int Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ k * Width + tx ];
		Pvalue += Mdelement * Ndelement;
	}
	Pd[ ty * Width + tx  ] = Pvalue; 
}
//END of kernel multiplication
*/

/*
END of KERNEL functions
*/


//Start of function to write Matrix to a text file
void writeMatFile( char *filename, int *matrix, int Width ) {
	FILE *fp;
	fp = fopen( filename, "w" );
	//print dummy file data headers for now
	fprintf( fp, "0 0");
	int x = 0;
	while( x < Width * Width ){
		fprintf( fp, " %d", matrix[ x ] );
		x++;		
	}
	fclose( fp );
//	printf( "\nMatrix was successfully written to filename: %s\n", filename );
}

//Start of function to print matrix
void printMatrix ( int *M, int rows, int columns ){
	//assumes matrix is in row-major format
	int index;
	printf ( "\n \n " );
	for ( int v = 0; v < rows; v++  ){
	//assumes a square matrix
		for ( int w = 0; w < columns; w++   ) {
			index = v * columns + w;
			printf ( " %02d", M[ index ]  );
		}
		printf ( " \n\n " );
	}
}//End of printMatrix function


//START of loadMatrixFile
void loadMatrixFile( char *filename, int *z, int matWidth, int matHeight ){
	int y = 0;
	int w = 0;
	int x;
	int offset = 0;
	FILE *ptr1 = fopen( filename, "r" );
	fscanf( ptr1, " %d", &x  );
	while( !feof( ptr1 ) && y < ( matWidth * matHeight ) + 1 ){
		if ( y > offset ){
			fscanf( ptr1, " %d", &z[ w - offset ]  );
			w++;
		}
		else{
			fscanf( ptr1, " %d", &x );
		}
		y++;
	}
	fclose( ptr1 );
}
//END of loadMatrixFile


//Start of matrix multiplication host function MatrixMul
//prototype: MatrixMul( confVec, spikVec, spikTransMat, width );
void MatrixMul( char *filename0, char *filename1, char *filename2, int Width, char *Cnext ){
	int size = Width * Width * sizeof( int );
	int *Md, *Nd, *Od, *Pd, *Qd;
	char outFile[ 20 ];

//	dim3 dimBlock( Width, Width );
//	dim3 dimGrid( 1, 1 );

	int *matA = ( int * )malloc( size );//spikVec
	loadMatrixFile( filename1, matA, Width, Width );
	printf( " \n%s after loading from file: \n", filename1 );
	printMatrix( matA, Width, Width );
			
	int *matB = ( int * )malloc( size );//spikTransMat
	loadMatrixFile( filename2, matB, Width, Width );		
	printf( " \n%s after loading from file: \n", filename2 );
	printMatrix( matB, Width, Width );

	int *matD = ( int * )malloc( size );//confVec
	loadMatrixFile( filename0, matD, Width, Width );		
	printf( " \n%s after loading from file: \n", filename0 );
	printMatrix( matD, Width, Width );
			
	//assumes a square matrix
	int *matC = ( int * )malloc( size );
	int *matE = ( int * )malloc( size );
	
//	cudaMalloc( ( void** ) &Md, size );//spikVec
//	cudaMemcpy( Md, matA, size, cudaMemcpyHostToDevice );

//	cudaMalloc( ( void** ) &Nd, size );//spikTransMat
//        cudaMemcpy( Nd, matB, size, cudaMemcpyHostToDevice );

	//Ck = spikVec * spikTransMat
//	cudaMalloc( ( void** ) &Pd, size );	

//	cudaMalloc( ( void** ) &Od, size );//confVec	

//        cudaMemcpy( Od, matD, size, cudaMemcpyHostToDevice );
	
	// final matrix: Ck+1 = confVec + Ck
//	cudaMalloc( ( void** ) &Qd, size );
	
	// Ck = spikVec * spikTransMat => Pd = Md * Nd
//	MatrixMulKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );
	MatrixMultiplication( matA, matB, matC, Width);

//	cudaMemcpy( matE, Qd, size, cudaMemcpyDeviceToHost );
	printf( " \n%s * %s : \n", filename1, filename2 );
	printMatrix( matC, Width, Width );

	// Ck+1 = confVec + Ck => Qd = Od + Pd
//	MatrixAddKernel<<< dimGrid, dimBlock >>>( Od, Pd, Qd, Width );
	MatrixAddKernel( matD, matC, matE, Width );

//	cudaMemcpy( matE, Qd, size, cudaMemcpyDeviceToHost );
	printf( " \n%s + %s * %s : \n", filename0, filename1, filename2 );
	printMatrix( matE, Width, Width );

	writeMatFile( Cnext, matE, Width );

	free( matA ); free( matB ); free( matC ); free( matD ); free( matE );
//	cudaFree( Md ); cudaFree( Nd ); cudaFree ( Pd ); cudaFree( Od ); cudaFree( Qd );
}
//End of Matrix multiplication function MatrixMul


/***
****END of AUXILIARY functions
****/




/***
****START of MAIN function
****/
int main ( int argc, char *argv[ ] ) {
	if ( argc < 6 ){
		printf( "\n Format: %s configurationVector spikingVector spikingTransitionMatrix squareMatrixWidth\n", argv[ 0 ] );
		exit( 1 );
	}
	char *confVec = argv[ 1 ];
	char *spikVec = argv[ 2 ];
	char *spikTransMat = argv[ 3 ];
	int width = atoi( argv[ 4 ] );
	char *Cnext = argv[ 5 ];
		
	if( ( strlen( confVec ) ) > 20 && ( strlen( spikVec ) ) > 20 && ( strlen( spikTransMat ) ) > 20  ) {
					printf( " Filename/s was/were too long ( > 20 char )  " );
					// TODO: Do something about segmentation fault here when input filename is > 20 chars
					//spikVec = { "\0" }; // doesn't work
					//*confVec = NULL; // doesn't work
	}
	else {
	//				printf( " You entered the file %s for the spiking vector \n", spikVec );
	//				printf( " You entered the file %s for the configuration vector \n", confVec );
	//				printf( " You entered the file %s for the spiking transition matrix \n ", spikTransMat );
		
	//				printf( "\nYou have entered files %s, %s, and %s and square matrix width %d \n", spikVec, confVec, spikTransMat, width );

					//load matrices from files
					FILE *ptr1 = fopen( confVec, "r" );
					FILE *ptr2 = fopen( spikVec, "r" );
					FILE *ptr3 = fopen( spikTransMat, "r" );

					if ( ptr1 == 0 && ptr2 == 0 && ptr3 == 0 ) {
						printf( "\n could not open one of the following files: %s %s %s \n", spikVec, confVec, spikTransMat );
						//should return something here
					}
					else {
						MatrixMul( confVec, spikVec, spikTransMat, width, Cnext );
					}
					fclose( ptr1 ); fclose( ptr2 ); fclose( ptr3 );
	}
}
/***
****END of MAIN function
***/

