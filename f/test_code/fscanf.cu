
#include <hip/hip_runtime.h>
#include <stdio.h>

int main ( int argc, char *argv[ ] ) {
	int x;

    	if ( argc != 2 ) /* argc should be 4 for correct execution */
    	{
		/* We print argv[0] assuming it is the program name */
		printf( "\nusage: %s filenametoread \n\n", argv[0] );
	}
	else
	{	//assumes space separate integer values e.g. -1 23 4 -56 6 77
		FILE *ptr = fopen( argv[ 1 ], "r" );
		if ( ptr == 0 )
			printf( "\n could not open file %s \n", argv[ 1 ] );
		else
		{
			fscanf( ptr, "%d", &x  );
			while( !feof( ptr ) ){
				printf( "\n %d \n", x );
				fscanf( ptr, "%d", &x );
			}
		}
	fclose( ptr );
	}
}
