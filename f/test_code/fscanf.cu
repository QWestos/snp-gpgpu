
#include <hip/hip_runtime.h>
#include <stdio.h>

int main ( int argc, char *argv[ ] ) {
	int x, y, *z;

    	if ( argc != 2 ) /* argc should be 4 for correct execution */
    	{
		/* We print argv[0] assuming it is the program name */
		printf( "\nusage: %s filenametoread \n\n", argv[0] );
	}
	else
	{	//assumes space separate integer values e.g. -1 23 4 -56 6 77
		FILE *ptr = fopen( argv[ 1 ], "r" );
		if ( ptr == 0 )
			printf( "\n could not open file %s \n", argv[ 1 ] );
		else
		{	
			y = 1;
			//int w = 0;
			fscanf( ptr, "%d", &x  );
			while( !feof( ptr ) ){
				if ( y < 3 ){
					fscanf( ptr, "%d", &x );
					printf( "\n A: y: %d MatEl: %d \n", y, x );
				}
				else {
					printf( "\n B: y: %d MatEl: %d ", y, x );
					fscanf( ptr, "%d", &z[ y - 3 ]  );
					printf( " z[ w ]: %d \n", z[ y - 3 ] );
					//w++;
					//z[ y - 3 ] = x;
				}
				y++;
			}
		}
	fclose( ptr );
	}
}
