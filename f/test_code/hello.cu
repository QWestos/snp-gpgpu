#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "common/book.h"
/*
To summarize, host pointers can access memory from host code, and device pointers can access memory from
device code.
You can pass pointers allocated with hipMalloc() to functions that
execute on the device.
You can use pointers allocated with hipMalloc()to read or write
memory from code that executes on the device.
You can pass pointers allocated with hipMalloc()to functions that
execute on the host.
You cannot use pointers allocated with hipMalloc()to read or write
memory from code that executes on the host.
*/

//This kernel function will run in the device
__global__ void add ( int a, int b, int *c ) {
	*c = a + b;
}

int main ( void  ) {
	int  c;
	int *dev_c;
	
	//pointer to a pointer and sizeof
	HANDLE_ERROR( hipMalloc ( (void**) &dev_c, sizeof(int) ) );

	//kernel call
	add<<<1,1>>>( 5, 26, dev_c);

	HANDLE_ERROR( hipMemcpy ( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost ) );
	printf( "5 + 26 = %d\n", c);
	hipFree (dev_c);

	return 0;
}
