
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <cuda.h>
//#include "cuda_runtime_api.h"
//#include <stdint.h>
//#include <stdlib.h>

//This is the working matrix multiplication code - very basic
/*
Done:
Done:
- printing of matrix in a more pleasant manner using printMatrix function
- command line arguments
- opens matrix files and reads the matrix successfully
- working array passing from main to auxiliary (loadMatrixFile) function :)
- fixed printing of matrix
- fixed erroneous matrix values by moving loading into host matrix multiplication function!

Problems:
- (fixed)  MatA and MatB values are overlapping and erroneous
*/


// START of Auxiliary functions

//Start of kernel multiplication
__global__ void MatrixMulKernel ( int  *Md, int *Nd, int *Pd, int Width ){
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ k * Width + tx ];
		Pvalue += Mdelement * Ndelement;
	}

	Pd[ ty * Width + tx  ] = Pvalue;
}
//End of kernel multiplication


//function to print matrix
void printMatrix ( int *M, int rows, int columns ){
	//assumes matrix is in row-major format
	int index;
	printf ( "\n \n " );
	for ( int v = 0; v < rows; v++  ){
	//assumes a square matrix
		for ( int w = 0; w < columns; w++   ) {
			index = v * columns + w;
			printf ( " [%d] %03d", index, M[ index ]  );
		}
		printf ( " \n\n " );
	}
}//End of printMatrix function

//Start of getMatWidth => Get width i.e. # of columns
int getMatWidth( char *filename ){
	int width;
	//assumes space separate integer values e.g. -1 23 4 -56 6 77
	//assumes first integer in file is row, 2nd integer is column
	FILE *ptr = fopen( filename, "r" );
	if ( ptr == 0 ){
		printf( "\n could not open file %s \n", filename );
		width = 0;
	}
	else{
		fscanf( ptr, "%d", &width  );
	}
	fclose( ptr );
	return width;
}//end of getMatWidth function


//Start of getMatHeight => Get height i.e. # of rows
int getMatHeight( char *filename ){
	int height, dummy;
	//assumes space separate integer values e.g. -1 23 4 -56 6 77
	//assumes first integer in file is row, 2nd integer is column
	FILE *ptr = fopen( filename, "r" );
	if ( ptr == 0 ){
		printf( "\n could not open file %s \n", filename );
		height = 0;
	}
	else{
		for ( int count = 1; count < 3; count++ ){
			if ( count != 2 )
				fscanf( ptr, "%d", &dummy );
		else
			fscanf( ptr, "%d", &dummy  );
			height = dummy;
		}
	}
	fclose( ptr );
	return height;
}//end of getMatHeight function


//START of loadMatrixFile
void loadMatrixFile( char *filename, int *z, int matWidth, int matHeight ){
	int y = 0;
	int w = 0;
	int x;
	int offset = 0;
	FILE *ptr1 = fopen( filename, "r" );
//	int *z = ( int * )malloc( sizeof( ( matWidth * matHeight ) ) );
	//int z[ ( matWidth * matHeight ) + offset ] ;
	fscanf( ptr1, " %d", &x  );
	while( !feof( ptr1 ) && y < ( matWidth * matHeight ) + 1 ){
		if ( y > offset ){
			fscanf( ptr1, " %d", &z[ w - offset ]  );
			printf( " B: z[ %d ]: %d \n", w, z[ w - offset ] );
			w++;
		}
		else{
			fscanf( ptr1, " %d", &x );
		}
		y++;
	}
	fclose( ptr1 );
//	x = y = w = 0;
//	array = &z[ 0 ];
//	free( z );
}
//END of loadMatrixFile


//Start of matrix multiplication host function MatrixMul
void MatrixMul( char *filename1, char *filename2, int Width /*, int *M, int *N, int  *P, int Width*/ ){
	int size = Width * Width * sizeof( int );
	int *Md, *Nd, *Pd;

	dim3 dimBlock( Width, Width );
	dim3 dimGrid( 1, 1 );

			//load matrices from files
			//get heigh/rows and width/columns of matrices
/*			int matWidthA = getMatWidth ( filename1  );
			int matHeightA = getMatHeight ( filename1  );
			
			int matWidthB = getMatWidth ( filename2  );
			int matHeightB = getMatHeight ( filename2  ); */
			
			int *matA = ( int * )malloc( size );
			printf( "Width and height of Matrix A: %d %d and init values are\n", Width, Width );
			printMatrix( matA, Width, Width );
			loadMatrixFile( filename1, matA, Width, Width );

			printf( " \nMatrix A after loading from file: \n" );
			printMatrix( matA, Width, Width );
			
			int *matB = ( int * )malloc( size );
			printf( "Width and height of Matrix B: %d %d and init values are\n", Width, Width );
			printMatrix( matB, Width, Width );
			loadMatrixFile( filename2, matB, Width, Width );
		
			printf( " \nMatrix B after loading from file: \n" );
			printMatrix( matB, Width, Width );
			
			//assumes a square matrix
			int *matC = ( int * )malloc( size );
			
			printf( "A: \n" );
			for ( int w = 0; w < Width * Width + 10; w++ ){
			        printf( "%d: %d \n",w,  matA[ w ] );
			}
			printf( "\n B:\n" );
			for ( int v = 0; v < Width * Width + 10; v++ ){
			        printf( "%d: %d \n", v, matB[ v ] );
			}
			printf( "\n" );

			//MatrixMul( matA, matB, matC, Width );
			

			printf( " \nMatrix C initially: \n" );
			printMatrix( matC, Width, Width );


	hipMalloc( (void**) &Md, size );
	hipMemcpy( Md, matA, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Nd, size );
        hipMemcpy( Nd, matB, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Pd, size );	
	
	MatrixMulKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );

	hipMemcpy( matC, Pd, size, hipMemcpyDeviceToHost );

	printf( " C:\n" );
	for ( int w = 0; w < Width * Width; w++ ){
		printf( "\n" );
		printf( " %d: %d  ", w, matC[w] );
		printf( "\n" );
	}

			printf( " \nMatrix C finally: \n" );
			printMatrix( matC, Width, Width );

	free( matA ); free( matB ); free( matC );
	hipFree( Md ); hipFree( Nd ); hipFree ( Pd );
}
//End of Matrix multiplication function MatrixMul


//END of Auxiliary functions


//START of Main function
int main ( int argc, char *argv[ ] ) {
	int offset = 2;

	if ( argc != 4 ) {
		printf( "\nusage: %s matrixFile1 matrixFile2 squarematrixwidth\n\n", argv [ 0 ] );
	}
	else {
		char *filename1 = argv[ 1 ];
		char *filename2 = argv[ 2 ];
		int width = atoi( argv[ 3 ] );
		
		printf( "you have entered files %s and %s and square matrix width %d \n", filename1, filename2, width );

		//load matrices from files
		FILE *ptr1 = fopen( filename1, "r" );
		FILE *ptr2 = fopen( filename2, "r" );

		if ( ptr1 == 0 && ptr2 == 0 )
			printf( "\n could not open one of the following files: %s %s \n", argv[ 1 ], argv[ 2 ] );
		else {
			MatrixMul( filename1, filename2, width );
		}
		fclose( ptr1 ); fclose( ptr2 );
	}
}
//END of Main function
