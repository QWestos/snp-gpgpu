
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <cuda.h>
//#include "cuda_runtime_api.h"
//#include <stdint.h>
//#include <stdlib.h>

//This is the working matrix multiplication code - very basic
/*
Done:
Done:
- printing of matrix in a more pleasant manner using printMatrix function
- command line arguments
- opens matrix files and reads the matrix successfully
- working array passing from main to auxiliary (loadMatrixFile) function :)
- fixed printing of matrix

Problems:
- MatA and MatB values are overlapping and erroneous
*/


// START of Auxiliary functions

//Start of kernel multiplication
__global__ void MatrixMulKernel ( int  *Md, int *Nd, int *Pd, int Width ){
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ k * Width + tx ];
		Pvalue += Mdelement * Ndelement;
	}

	Pd[ ty * Width + tx  ] = Pvalue;
}
//End of kernel multiplication


//function to print matrix
void printMatrix ( int *M, int rows, int columns ){
	//assumes matrix is in row-major format
	int index;
	printf ( "\n \n " );
	for ( int v = 0; v < rows; v++  ){
	//assumes a square matrix
		for ( int w = 0; w < columns; w++   ) {
			index = v * columns + w;
			printf ( " [%d] %03d", index, M[ index ]  );
		}
		printf ( " \n\n " );
	}
}//End of printMatrix function


//Start of matrix multiplication host function
void MatrixMul( int *M, int *N, int  *P, int Width ){
	int size = Width * Width * sizeof( int );
	int *Md, *Nd, *Pd;

	hipMalloc( (void**) &Md, size );
	hipMemcpy( Md, M, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Nd, size );
        hipMemcpy( Nd, N, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Pd, size );
	
	dim3 dimBlock( Width, Width );
	dim3 dimGrid( 1, 1 );
	
	MatrixMulKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );

	hipMemcpy( P, Pd, size, hipMemcpyDeviceToHost );

	printf( " C:\n" );
	for ( int w = 0; w < Width * Width; w++ ){
		printf( "\n" );
		printf( " %d: %d  ", w, P[w] );
		printf( "\n" );
	}

	//printMatrix( P, Width, Width );

	hipFree( Md ); hipFree( Nd ); hipFree ( Pd );
}
//End of Matrix multiplication function


//Start of getMatWidth => Get width i.e. # of columns
int getMatWidth( char *filename ){
	int width;
	//assumes space separate integer values e.g. -1 23 4 -56 6 77
	//assumes first integer in file is row, 2nd integer is column
	FILE *ptr = fopen( filename, "r" );
	if ( ptr == 0 ){
		printf( "\n could not open file %s \n", filename );
		width = 0;
	}
	else{
		fscanf( ptr, "%d", &width  );
	}
	fclose( ptr );
	return width;
}//end of getMatWidth function


//Start of getMatHeight => Get height i.e. # of rows
int getMatHeight( char *filename ){
	int height, dummy;
	//assumes space separate integer values e.g. -1 23 4 -56 6 77
	//assumes first integer in file is row, 2nd integer is column
	FILE *ptr = fopen( filename, "r" );
	if ( ptr == 0 ){
		printf( "\n could not open file %s \n", filename );
		height = 0;
	}
	else{
		for ( int count = 1; count < 3; count++ ){
			if ( count != 2 )
				fscanf( ptr, "%d", &dummy );
		else
			fscanf( ptr, "%d", &dummy  );
			height = dummy;
		}
	}
	fclose( ptr );
	return height;
}//end of getMatHeight function


//START of loadMatrixFile
void loadMatrixFile( char *filename, int *z, int matWidth, int matHeight ){
	int y = 0;
	int w = 0;
	int x;
	int offset = 0;
	FILE *ptr1 = fopen( filename, "r" );
//	int *z = ( int * )malloc( sizeof( ( matWidth * matHeight ) ) );
	//int z[ ( matWidth * matHeight ) + offset ] ;
	fscanf( ptr1, " %d", &x  );
	while( !feof( ptr1 ) && y < ( matWidth * matHeight ) + 1 ){
		if ( y > offset ){
			fscanf( ptr1, " %d", &z[ w - offset ]  );
			printf( " B: z[ %d ]: %d \n", w, z[ w - offset ] );
			w++;
		}
		else{
			fscanf( ptr1, " %d", &x );
		}
		y++;
	}
	fclose( ptr1 );
//	x = y = w = 0;
//	array = &z[ 0 ];
//	free( z );
}
//END of loadMatrixFile

//END of Auxiliary functions


//START of Main function
int main ( int argc, char *argv[ ] ) {
	int offset = 2;

	if ( argc != 3 ) {
		printf( "\nusage: %s matrixFile1 matrixFile2 \n\n", argv [ 0 ] );
	}
	else {
		char *filename1 = argv[ 1 ];
		char *filename2 = argv[ 2 ];
		//int *matA; //holds 1st matrix
		//int *matB; //holds 2nd matrix
		
		printf( "you have entered files %s and %s \n", filename1, filename2 );
		//load matrices from files
		FILE *ptr1 = fopen( filename1, "r" );
		FILE *ptr2 = fopen( filename2, "r" );

		if ( ptr1 == 0 && ptr2 == 0 )
			printf( "\n could not open one of the following files: %s %s \n", argv[ 1 ], argv[ 2 ] );
		else {
		//load matrices from files
			//get heigh/rows and width/columns of matrices
			int matWidthA = getMatWidth ( filename1  );
			int matHeightA = getMatHeight ( filename1  );
			
			int matWidthB = getMatWidth ( filename2  );
			int matHeightB = getMatHeight ( filename2  );
			
			int *matA = ( int * )malloc( sizeof( int ) * matWidthA );
			printf( "Width and height of Matrix A: %d %d and init values are\n", matWidthA, matWidthA );
			printMatrix( matA, matHeightA, matWidthA );
			loadMatrixFile( filename1, matA, matWidthA, matHeightA );

			printf( " \nMatrix A after loading from file: \n" );
			printMatrix( matA, matHeightA, matWidthA );
			
			int *matB = ( int * )malloc( sizeof( int ) * matWidthB );
			printf( "Width and height of Matrix B: %d %d and init values are\n", matWidthB, matWidthB );
			printMatrix( matB, matHeightB, matWidthB );
			loadMatrixFile( filename2, matB, matWidthB, matHeightB );
		
			printf( " \nMatrix A after loading from file: \n" );
			printMatrix( matA, matHeightA, matWidthA );
			
			printf( " \nMatrix B after loading from file: \n" );
			printMatrix( matB, matHeightB, matWidthB );

			//assumes a square matrix
			int *matC = ( int * )malloc( sizeof( int ) * matWidthA );
			

			printf( "A: \n" );
			for ( int w = 0; w < matWidthA * matWidthA + 10; w++ ){
			        printf( "%d: %d \n",w,  matA[ w ] );
			}
			printf( "\n B:\n" );
			for ( int v = 0; v < matWidthB * matWidthB + 10; v++ ){
			        printf( "%d: %d \n", v, matB[ v ] );
			}
			printf( "\n" );

			free( matA ); free( matB ); free( matC );
			
			printf( "A: \n" );
			for ( int w = 0; w < matWidthA * matWidthA + 10; w++ ){
			        printf( "%d: %d \n",w,  matA[ w ] );
			}
			printf( "\n B:\n" );
			for ( int v = 0; v < matWidthB * matWidthB + 10; v++ ){
			        printf( "%d: %d \n", v, matB[ v ] );
			}
			printf( "\n" );
			
			MatrixMul( matA, matB, matC, matWidthA );
			
			printf( " \nMatrix C: \n" );
			printMatrix( matC, matWidthA, matWidthA );

			free( matA ); free( matB ); free( matC );
		}
		fclose( ptr1 ); fclose( ptr2 );
	}
}
//END of Main function
