
#include <hip/hip_runtime.h>
#include <stdio.h>

int main ( void ) {
	int x;
	while( x != 3) {
		printf( "\n Type \n 1 to enter filenames < 20 in length \n 2 for 2 \n 3 to quit \n: " );
		scanf( "%d", &x );
		switch( x ){
			case 1:
				char spikVec[ 20 ], confVec[ 20 ], spikTransMat[ 20 ];

				printf( " Please enter spiking vector file: \n" );
				scanf( " %s", &spikVec );
				printf( " Please enter configuration vector file: \n" );
				scanf( " %s", &confVec );
				printf( " Please enter spiking transition matrix file: \n" );
				scanf( " %s", &spikTransMat ); 
				if( ( strlen( spikVec ) ) > 20 && ( strlen( confVec ) ) > 20 && ( strlen( spikTransMat ) ) > 20  ) {
					printf( " Filename/s was/were too long ( > 20 char )  " );
					// Do something about segmentation fault here
					//spikVec = { "\0" }; // doesn't work
					//*confVec = NULL; // doesn't work 
				}
				else {
					printf( " You entered the file %s for the spiking vector \n", spikVec );
					printf( " You entered the file %s for the configuration vector \n", confVec );
					printf( " You entered the file %s for the spiking transition matrix \n ", spikTransMat );
				}
				break;
			case 2: 
				printf( " You entered 2 \n" );
				break;
			case 3: 
				printf( " You entered quit. Bye! \n" );
				break;
			default:
				printf( " You entered an invalid choice \n" );
				break;
		}

	}
}
