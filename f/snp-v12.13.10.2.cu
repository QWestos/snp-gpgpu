
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <cuda.h>
//#include "cuda_runtime_api.h"
//#include <stdint.h>
//#include <stdlib.h>

//This is the working matrix multiplication code - very basic
/****

Done:
- printing of matrix in a more pleasant manner using printMatrix function
- command line arguments
- opens matrix files and reads the matrix successfully
- working array passing from main to auxiliary (loadMatrixFile) function :)
- fixed printing of matrix
- fixed erroneous matrix values by moving loading into host matrix multiplication function!
- basic move towards SN P simulation: multiplication of s0 and Msnp
- moving from multiplication to finally simulating an SN P (sort of) in a very basic manner
- MatrixAddKernel now works :)

Problems:
- (fixed)  MatA and MatB values are overlapping and erroneous

TODOS:
- error checking of switch case input ( scanf of int and char )
- use multiple files + make file
- see code comments

****/


/***
**** START of AUXILIARY functions
***/

/*
START of KERNEL functions
*/
//START vector addition kernel function
__global__ void MatrixAddKernel ( int  *Md, int *Nd, int *Pd, int Width ){
	// MatrixAddKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );
	//dim3 dimBlock( Width, Width ); dim3 dimGrid( 1, 1 );
	//int tx = threadIdx.x;
	int ty = threadIdx.y;
	//due to row-major ordering of matrix elements
	//int Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ ty * Width + k ];
		Pd[ ty * Width + k ] = Mdelement + Ndelement;
	}
	//Pd[ ty * Width + tx  ] = Pvalue;
}							
//END of kernel addition


//START of kernel multiplication
__global__ void MatrixMulKernel ( int  *Md, int *Nd, int *Pd, int Width ){
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	//due to row-major ordering of matrix elements
	int Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ k * Width + tx ];
		Pvalue += Mdelement * Ndelement;
	}
	Pd[ ty * Width + tx  ] = Pvalue; 
}
//END of kernel multiplication


/*
END of KERNEL functions
*/


//function to print matrix
void printMatrix ( int *M, int rows, int columns ){
	//assumes matrix is in row-major format
	int index;
	printf ( "\n \n " );
	for ( int v = 0; v < rows; v++  ){
	//assumes a square matrix
		for ( int w = 0; w < columns; w++   ) {
			index = v * columns + w;
			printf ( " %02d", M[ index ]  );
		}
		printf ( " \n\n " );
	}
}//End of printMatrix function


//START of loadMatrixFile
void loadMatrixFile( char *filename, int *z, int matWidth, int matHeight ){
	int y = 0;
	int w = 0;
	int x;
	int offset = 0;
	FILE *ptr1 = fopen( filename, "r" );
	fscanf( ptr1, " %d", &x  );
	while( !feof( ptr1 ) && y < ( matWidth * matHeight ) + 1 ){
		if ( y > offset ){
			fscanf( ptr1, " %d", &z[ w - offset ]  );
			w++;
		}
		else{
			fscanf( ptr1, " %d", &x );
		}
		y++;
	}
	fclose( ptr1 );
}
//END of loadMatrixFile


//Start of matrix multiplication host function MatrixMul
//prototype: MatrixMul( confVec, spikVec, spikTransMat, width );
void MatrixMul( char *filename0, char *filename1, char *filename2, int Width ){
	int size = Width * Width * sizeof( int );
	int *Md, *Nd, *Od, *Pd, *Qd;

	dim3 dimBlock( Width, Width );
	dim3 dimGrid( 1, 1 );

	int *matA = ( int * )malloc( size );//spikVec
	loadMatrixFile( filename1, matA, Width, Width );
	printf( " \n%s after loading from file: \n", filename1 );
	printMatrix( matA, Width, Width );
			
	int *matB = ( int * )malloc( size );//spikTransMat
	loadMatrixFile( filename2, matB, Width, Width );		
	printf( " \n%s after loading from file: \n", filename2 );
	printMatrix( matB, Width, Width );

	int *matD = ( int * )malloc( size );//confVec
	loadMatrixFile( filename0, matD, Width, Width );		
	printf( " \n%s after loading from file: \n", filename0 );
	printMatrix( matD, Width, Width );
			
	//assumes a square matrix
	int *matC = ( int * )malloc( size );
	int *matE = ( int * )malloc( size );
	
	hipMalloc( ( void** ) &Md, size );//spikVec
	hipMemcpy( Md, matA, size, hipMemcpyHostToDevice );

	hipMalloc( ( void** ) &Nd, size );//spikTransMat
        hipMemcpy( Nd, matB, size, hipMemcpyHostToDevice );

	//Ck = spikVec * spikTransMat
	hipMalloc( ( void** ) &Pd, size );	

	hipMalloc( ( void** ) &Od, size );//confVec	

        hipMemcpy( Od, matD, size, hipMemcpyHostToDevice );
	
	// final matrix: Ck+1 = confVec + Ck
	hipMalloc( ( void** ) &Qd, size );
	
	// Ck = spikVec * spikTransMat => Pd = Md * Nd
	MatrixMulKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );

//	cudaMemcpy( matE, Qd, size, cudaMemcpyDeviceToHost );
//	printf( " \n%s * %s : \n", filename1, filename2 );
//	printMatrix( matC, Width, Width );

	// Ck+1 = confVec + Ck => Qd = Od + Pd
	MatrixAddKernel<<< dimGrid, dimBlock >>>( Od, Pd, Qd, Width );

	hipMemcpy( matE, Qd, size, hipMemcpyDeviceToHost );
	printf( " \n%s + %s * %s : \n", filename0, filename1, filename2 );
	printMatrix( matE, Width, Width );

	free( matA ); free( matB ); free( matC ); free( matD ); free( matE );
	hipFree( Md ); hipFree( Nd ); hipFree ( Pd ); hipFree( Od ); hipFree( Qd );
}
//End of Matrix multiplication function MatrixMul


/***
****END of AUXILIARY functions
****/




/***
****START of MAIN function
****/
int main ( void ) {
	int x;
	while( x != 2 ) {
		printf( "\n Type \n 1 to enter filenames < 20 in length \n 2 to quit \n: " );
		scanf( "%d", &x );
		switch( x ){
			case 1:
				char a[ 20 ], b[ 20 ], c[ 20 ];
				int d;
				printf( " Please enter spiking vector file: \n" );
				scanf( " %s", &a );
				printf( " Please enter configuration vector file: \n" );
				scanf( " %s", &b );
				printf( " Please enter spiking transition matrix file: \n" );
				scanf( " %s", &c ); 
				printf( " Please enter the square matrix width: \n" );
				scanf( " %d", &d ); 

				if( ( strlen( a ) ) > 20 && ( strlen( b ) ) > 20 && ( strlen( c ) ) > 20  ) {
					printf( " Filename/s was/were too long ( > 20 char )  " );
					// TODO: Do something about segmentation fault here when input filename is > 20 chars
					//spikVec = { "\0" }; // doesn't work
					//*confVec = NULL; // doesn't work
					break;
				}
				else {
					printf( " You entered the file %s for the spiking vector \n", a );
					printf( " You entered the file %s for the configuration vector \n", b );
					printf( " You entered the file %s for the spiking transition matrix \n ", c );
					char *confVec = b;
					char *spikVec = a;
					char *spikTransMat = c;
					int width = d;
		
					printf( "\nYou have entered files %s, %s, and %s and square matrix width %d \n", spikVec, confVec, spikTransMat, width );

					//load matrices from files
					FILE *ptr1 = fopen( confVec, "r" );
					FILE *ptr2 = fopen( spikVec, "r" );
					FILE *ptr3 = fopen( spikTransMat, "r" );

					if ( ptr1 == 0 && ptr2 == 0 && ptr3 == 0 ) {
						printf( "\n could not open one of the following files: %s %s %s \n", a, b, c );
						break;
					}
					else {
						MatrixMul( confVec, spikVec, spikTransMat, width );
					}
					fclose( ptr1 ); fclose( ptr2 ); fclose( ptr3 );
					break;				
				}
			case 2: 
				printf( " You entered quit. Bye! \n\n" );
				break;
			default:
				printf( " You entered an invalid choice \n\n" );
				break;
		}
	}
}
/***
****END of MAIN function
***/
