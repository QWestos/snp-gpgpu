
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <cuda.h>
//#include "cuda_runtime_api.h"
//#include <stdint.h>
//#include <stdlib.h>

//#define Width 4

/*
** START of auxiliary functions
*/

//Matrix multiplication kernel function
__global__ void MatrixMulKernel ( int  *Md, int *Nd, int *Pd, int Width ){
	//2D thread ID
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//Pvalue stores Pd element computed by thread
	int Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ k * Width + tx ];
		Pvalue += Mdelement * Ndelement;
	}

	//Write matrix to device memory; each thread writes one element
	Pd[ ty * Width + tx  ] = Pvalue;
}// End of Matrix multiplication kernel function

//function to print matrix
void printMatrix ( int *M, int rows, int columns ){
	//assumes matrix is in row-major format
	printf ( "\n %s: \n", "M" );
	for ( int v = 0; v < rows; v++  ){
	//assumes a square matrix
		for ( int w = 0; w < columns; w++   ) {
			printf ( " %03d ", M[ v * columns + w ]  );
		}
		printf ( " \n " );
	}
}//End of printMatrix function

//Matrix multiplication function
// assumes a SQUARE matrix for now
void MatrixMul( int *M, int *N, int *P, int Width ){
	int size = Width * Width * sizeof( int );
	int *Md, *Nd, *Pd;

	//Transfer M, N to device
	hipMalloc( (void**) &Md, size );
	hipMemcpy( Md, M, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Nd, size );
        hipMemcpy( Nd, N, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Pd, size );
	
	//invoke kernel
	dim3 dimBlock( Width, Width );
	dim3 dimGrid( 1, 1 );
	
	//Launch kernel
	MatrixMulKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );

	//transfer from device to host
	hipMemcpy( P, Pd, size, hipMemcpyDeviceToHost );
/*
	//Print matrix P
	for ( int w = 0; w < Width * Width; w++ ){
		printf( "\n" );
		printf( " %d: %d  ", w, P[w] );
		printf( "\n" );
	}

	printMatrix( P, 4, 4 ); */

	//Free device matrices
	hipFree( Md ); hipFree( Nd ); hipFree ( Pd );
}//End of MatrixMul function	

//Start of getMatWidth => Get width i.e. # of columns
int getMatWidth( char *filename ){
	int width;
	//assumes space separate integer values e.g. -1 23 4 -56 6 77
	//assumes first integer in file is row, 2nd integer is column
	FILE *ptr = fopen( filename, "r" );
	if ( ptr == 0 ){
		printf( "\n could not open file %s \n", filename );
		width = 0;
	}
	else{
		fscanf( ptr, "%d", &width  );
	}
	fclose( ptr );
	return width;
}//end of getMatWidth function

//Start of getMatHeight => Get height i.e. # of rows
int getMatHeight( char *filename ){
	int height, dummy;
	//assumes space separate integer values e.g. -1 23 4 -56 6 77
	//assumes first integer in file is row, 2nd integer is column
	FILE *ptr = fopen( filename, "r" );
	if ( ptr == 0 ){
		printf( "\n could not open file %s \n", filename );
		height = 0;
	}
	else{
		for ( int count = 1; count < 3; count++ ){
			if ( count != 2 )
				fscanf( ptr, "%d", &dummy );
			else
				fscanf( ptr, "%d", &dummy  );
				height = dummy;
		}
	}
	fclose( ptr );
	return height;
}//end of getMatHeight function

//START of loadMatrixFile function
void loadMatrixFile( char *filename, int *array, int cols, int rows ) {
	//assumes space separate integer values e.g. -1 23 4 -56 6 77
	int x, y, *dummy;
	FILE *matFile = fopen( filename, "r" );
	if ( matFile == 0 ){
		printf( "\n could not open file %s \n", filename );
	}
	else{
		y = 1;
		int offset = 4;
		//z = 0;
		fscanf( matFile, "%d", &x );
		while( !feof( matFile ) && y <  rows * cols + offset ) {
			if ( y < offset ){
				fscanf( matFile, "%d", &x );
				printf( " A: y = %d x = %d \n ", y, x );
			}
			else {
				fscanf( matFile, "%d", &dummy[ y - offset ] );
				//fscanf( matFile, "%d", &x );
				//printf( " B: y = %d x = %d \n", y, x );
				printf( " B: y = %d dummy[ z ] = %d \n", y, dummy[ y - offset ] );
				//z++;
				//array[ y - offset ] = x;
			}
			y++;
		} 
	}
	fclose( matFile ); 
	//return array; 
}//END of loadMatrixFile function

/*
** END OF Auxiliary functions
*/


/*
** START OF MAIN FUNCTION
*/

int main ( int argc, char *argv[ ] ) {
	int Width = 4;
	
	//populate arrays to multiply
	int A[ Width * Width ];

	for ( int x = 0; x < Width * Width; x++ ){
		A[ x ] = 1;
	}
	
	int B[ Width * Width ];
	for ( int z = 0; z < Width * Width; z++ ){
		B[ z ] = 2;
	}
	
	int C[ Width * Width ];
	
	char *filename1 = argv[ 1 ];
	char *filename2 = argv[ 2 ];
	int *matA; //holds first matrix
	int *matB; //holds sencond matrix

	if ( argc != 3 ) /* argc should be 4 for correct execution */ {
		/* We print argv[0] assuming it is the program name */
		printf( "\nusage: %s matrixFile1 matrixFile2 \n\n", argv [0 ] );
	}
	else {

		//returns # of cols of matrix, zero otherwise
		int matWidthA = getMatWidth ( filename1  );
		//get # of rows of matrix, zero otherwise
		int matHeightA = getMatHeight( filename1 );

		//returns # of cols of matrix, zero otherwise
		int matWidthB = getMatWidth ( filename2  );
		//get # of rows of matrix, zero otherwise
		int matHeightB = getMatHeight( filename2 );
		
		//load matrices from files
		loadMatrixFile( filename1, matA, matWidthA, matHeightA );
		//loadMatrixFile( filename2, matB );

        //Print matrix P
	        for ( int w = 0; w < matWidthA * matWidthA; w++ ){
		        printf( "\n" );
			printf( " %d: %d  ", w, matA[ w ] );
			printf( "\n" );
		}
		//printMatrix( matA, matWidthA, matHeightA );

		//printf( " widht of matrix A: %d \n ", matWidthA );
		//printf( "height of matrix A: %d \n\n", matHeightA );
	}
	//MatrixMul( A, B, C, Width );
}
/*
** END OF MAIN FUNCTION
*/
