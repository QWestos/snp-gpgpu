#include "hip/hip_runtime.h"
//#include "book.h"
#include <stdio.h>

#define N 10

//kernel function to add 2 vectors. These functions can be called
// from the host/device, but will run on the device only
__global__ void add( int *a, int *b, int *c ) {
	int tid = blockIdx.x; //thred id
	if ( tid < N )
		c[tid] =a[tid] + b[tid];
}

int main( void ) {
	int /*a[N], b[N],*/ c[N];
	int *dev_a, *dev_b, *dev_c;
	// allocate the memory on the GPU
	HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );
	// fill the arrays 'a' and 'b' on the CPU
	int a[N] = { 1, 1, 2, 2, 4, 4, 5, 5, 6, 6  };
	int b[N] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10 };
	/*for (int i=0; i<N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}*/
	// copy the arrays 'a' and 'b' to the GPU
	HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice) );
	//call kernel function,run kernel function on N blocks 
	add<<<N, 1>>>( dev_a, dev_b, dev_c );
	//copy array 'c' from GPU to CPU for printing etc
	HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof( int), hipMemcpyDeviceToHost ) );
																				
																						//display results
																						for (int i = 0; i < N; i++){
		printf( "\n %d  + %d = %d\n ", a[i], b[i], c[i] );
	}
																						//free memory allocated on the GPU
		hipFree( dev_a );
		hipFree( dev_b );
		hipFree( dev_c );

	return 0;
																					}
