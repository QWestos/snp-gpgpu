#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <stdint.h>
#include <stdlib.h>

//This is the working matrix multiplication code - very basic
/*
Done:
- printing of matrix in a more pleasant manner using printMatrix function
- command line arguments
- opens matrix files and reads the matrix successfully
*/


// START of Auxiliary functions

//Start of kernel multiplication
__global__ void MatrixMulKernel ( int  *Md, int *Nd, int *Pd, int Width ){
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ k * Width + tx ];
		Pvalue += Mdelement * Ndelement;
	}

	Pd[ ty * Width + tx  ] = Pvalue;
}
//End of kernel multiplication

//Start of matrix multiplication host function
void MatrixMul( int *M, int *N, int  *P, int Width ){
	int size = Width * Width * sizeof( int );
	int *Md, *Nd, *Pd;

	hipMalloc( (void**) &Md, size );
	hipMemcpy( Md, M, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Nd, size );
        hipMemcpy( Nd, N, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Pd, size );
	
	dim3 dimBlock( Width, Width );
	dim3 dimGrid( 1, 1 );
	
	MatrixMulKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );

	hipMemcpy( P, Pd, size, hipMemcpyDeviceToHost );

	for ( int w = 0; w < Width * Width; w++ ){
		printf( "\n" );
		printf( " %d: %f  ", w, P[w] );
		printf( "\n" );
	}

	hipFree( Md ); hipFree( Nd ); hipFree ( Pd );
}
//End of Matrix multiplication function

//Start of getMatWidth => Get width i.e. # of columns
int getMatWidth( char *filename ){
	int width;
	//assumes space separate integer values e.g. -1 23 4 -56 6 77
	//assumes first integer in file is row, 2nd integer is column
	FILE *ptr = fopen( filename, "r" );
	if ( ptr == 0 ){
		printf( "\n could not open file %s \n", filename );
		width = 0;
	}
	else{
		fscanf( ptr, "%d", &width  );
	}
	fclose( ptr );
	return width;
}//end of getMatWidth function

//Start of getMatHeight => Get height i.e. # of rows
int getMatHeight( char *filename ){
	int height, dummy;
	//assumes space separate integer values e.g. -1 23 4 -56 6 77
	//assumes first integer in file is row, 2nd integer is column
	FILE *ptr = fopen( filename, "r" );
	if ( ptr == 0 ){
		printf( "\n could not open file %s \n", filename );
		height = 0;
	}
	else{
		for ( int count = 1; count < 3; count++ ){
			if ( count != 2 )
				fscanf( ptr, "%d", &dummy );
		else
			fscanf( ptr, "%d", &dummy  );
			height = dummy;
		}
	}
	fclose( ptr );
	return height;
}//end of getMatHeight function


//function to print matrix
void printMatrix ( int *M, int rows, int columns ){
	//assumes matrix is in row-major format
	printf ( "\n %s: \n", "M" );
	for ( int v = 0; v < rows; v++  ){
	//assumes a square matrix
		for ( int w = 0; w < columns; w++   ) {
			printf ( " %03d ", M[ v * columns + w ]  );
		}
		printf ( " \n " );
	}
}//End of printMatrix function

//END of Auxiliary functions


//START of Main function
int main ( int argc, char *argv[ ] ) {
	
	if ( argc != 3 ) {
		printf( "\nusage: %s matrixFile1 matrixFile2 \n\n", argv [ 0 ] );
	}
	else {
		char *filename1 = argv[ 1 ];
		char *filename2 = argv[ 2 ];
		int *matA; //holds 1st matrix
		int *matB; //holds 2nd matrix
		
		matA = ( int * ) malloc( sizeof ( int ) );
		matB = ( int * ) malloc( sizeof ( int ) );

		printf( "you have entered files %s and %s \n", filename1, filename2 );
		//load matrices from files
		FILE *ptr1 = fopen( filename1, "r" );
		FILE *ptr2 = fopen( filename2, "r" );

		if ( ptr1 == 0 && ptr2 == 0 )
			printf( "\n could not open one of the following files: %s %s \n", argv[ 1 ], argv[ 2 ] );
		else {
		//load matrices from files
			//get heigh/rows and width/columns of matrices
			int matWidthA = getMatWidth ( filename1  );
			int matHeightA = getMatHeight ( filename1  );
			
			int matWidthB = getMatWidth ( filename2  );
			int matHeightB = getMatHeight ( filename2  );

			int y = 1;
			int x;
			int offset = 2;
			int z[ ( matWidthA * matHeightA ) + offset ] ;
			fscanf( ptr1, " %d", &x  );
			while( !feof( ptr1 ) && y < ( matWidthA * matHeightA ) + offset ){
				if ( y > offset ){
                                   	fscanf( ptr1, " %d", &z[ y - offset ]  );
					printf( " B: z[ %d ]: %d \n", y, z[ y - offset ] );
					//fscanf( ptr1, " %d", &x );
					//printf( "\n A: y: %d MatEl: %d \n", y, x );
				} /*
				else{
					fscanf( ptr1, " %d", &z[ y - offset ]  );
					printf( " B: z[ %d ]: %d \n", y, z[ y - offset ] );
				} */
				y++;
			}
		}

		free( matA ); free( matB );
	}

	int Width = 4;
	
	int A[ Width * Width ];

	for ( int x = 0; x < Width * Width; x++ ){
		A[ x ] = 2;
	}
	
	int B[ Width * Width ];
	for ( int z = 0; z < Width * Width; z++ ){
		B[ z ] = 2;
	}
	
	int C[ Width * Width ];

	//MatrixMul( A, B, C, Width );
	//printMatrix( C, Width, Width );
}
//END of Main function
