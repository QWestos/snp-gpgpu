#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <stdint.h>
#include <stdlib.h>

//This is the working matrix multiplication code - very basic
/*
Done:
- printing of matrix in a more pleasant manner using printMatrix function
- getting commandline arguments
*/


// START of Auxiliary functions

//Start of kernel multiplication
__global__ void MatrixMulKernel ( int  *Md, int *Nd, int *Pd, int Width ){
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ k * Width + tx ];
		Pvalue += Mdelement * Ndelement;
	}

	Pd[ ty * Width + tx  ] = Pvalue;
}
//End of kernel multiplication

//Start of matrix multiplication host function
void MatrixMul( int *M, int *N, int  *P, int Width ){
	int size = Width * Width * sizeof( int );
	int *Md, *Nd, *Pd;

	hipMalloc( (void**) &Md, size );
	hipMemcpy( Md, M, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Nd, size );
        hipMemcpy( Nd, N, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Pd, size );
	
	dim3 dimBlock( Width, Width );
	dim3 dimGrid( 1, 1 );
	
	MatrixMulKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );

	hipMemcpy( P, Pd, size, hipMemcpyDeviceToHost );

	for ( int w = 0; w < Width * Width; w++ ){
		printf( "\n" );
		printf( " %d: %f  ", w, P[w] );
		printf( "\n" );
	}

	hipFree( Md ); hipFree( Nd ); hipFree ( Pd );
}
//End of Matrix multiplication function

//function to print matrix
void printMatrix ( int *M, int rows, int columns ){
	//assumes matrix is in row-major format
	printf ( "\n %s: \n", "M" );
	for ( int v = 0; v < rows; v++  ){
	//assumes a square matrix
		for ( int w = 0; w < columns; w++   ) {
			printf ( " %03d ", M[ v * columns + w ]  );
		}
		printf ( " \n " );
	}
}//End of printMatrix function

//END of Auxiliary functions


//START of Main function
int main ( int argc, char *argv[ ] ) {
	
	if ( argc != 3 ) {
		printf( "\nusage: %s matrixFile1 matrixFile2 \n\n", argv [ 0 ] );
	}
	else {
		char *filename1 = argv[ 1 ];
		char *filename2 = argv[ 2 ];
		int *matA; //holds 1st matrix
		int *matB; //holds 2nd matrix
		
		matA = ( int * ) malloc( sizeof ( int ) );
		matB = ( int * ) malloc( sizeof ( int ) );

		printf( "you have entered files %s and %s \n", filename1, filename2 );
		free( matA ); free( matB );
	}

	int Width = 4;
	
	int A[ Width * Width ];

	for ( int x = 0; x < Width * Width; x++ ){
		A[ x ] = 2;
	}
	
	int B[ Width * Width ];
	for ( int z = 0; z < Width * Width; z++ ){
		B[ z ] = 2;
	}
	
	int C[ Width * Width ];

	//MatrixMul( A, B, C, Width );
	//printMatrix( C, Width, Width );
}
//END of Main function
