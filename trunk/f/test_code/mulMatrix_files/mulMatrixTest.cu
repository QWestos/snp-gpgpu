
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <cuda.h>

/*
** START of auxiliary functions
*/

//Matrix multiplication kernel function
__global__ void MatrixMulKernel ( int  *Md, int *Nd, int *Pd, int Width ){
	//2D thread ID
		int tx = threadIdx.x;
			int ty = threadIdx.y;

				//Pvalue stores Pd element computed by thread
					int Pvalue = 0;
						for ( int k = 0; k < Width; ++k ){
								int Mdelement = Md[ ty * Width + k ];
										int Ndelement = Nd[ k * Width + tx ];
												Pvalue += Mdelement * Ndelement;
													}

														//Write matrix to device memory; each thread writes one element
															Pd[ ty * Width + tx  ] = Pvalue;
															}// End of Matrix multiplication kernel function

															//function to print matrix
															void printMatrix ( int *M, int rows, int columns ){
																//assumes matrix is in row-major format
																	printf ( "\n %s: \n", "M" );
																		for ( int v = 0; v < rows; v++  ){
																				//assumes a square matrix
																						for ( int w = 0; w < columns; w++   ) {
																								printf ( " %03d ", M[ v * columns + w ]  );
																										}
																												printf ( " \n " );
																													}
																													}//End of printMatrix function

																													//Matrix multiplication function
																													// assumes a SQUARE matrix for now
																													void MatrixMul( int *M, int *N, int *P, int Width ){
																														int size = Width * Width * sizeof( int );
																															int *Md = NULL;
																																int *Nd = NULL;
																																	int *Pd = NULL;

																																		//Transfer M, N to device
																																			hipMalloc( (void**) &Md, size );
																																				hipMemcpy( Md, M, size, hipMemcpyHostToDevice );
																																					hipMalloc( (void**) &Nd, size );
																																					        hipMemcpy( Nd, N, size, hipMemcpyHostToDevice );
																																							hipMalloc( (void**) &Pd, size );
																																								
																																									//invoke kernel
																																										dim3 dimBlock( Width, Width );
																																											dim3 dimGrid( 1, 1 );
																																												
																																													//Launch kernel
																																														MatrixMulKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );

																																															//transfer from device to host
																																																hipMemcpy( P, Pd, size, hipMemcpyDeviceToHost );

																																																	//Print matrix P
																																																	/*	for ( int w = 0; w < Width * Width; w++ ){
																																																			printf( "\n" );
																																																					printf( " %d: %d  ", w, P[ w ] );
																																																							printf( "\n" );
																																																								} */

																																																									//printMatrix( P, 4, 4 );

																																																										//Free device matrices
																																																											hipFree( Md ); hipFree( Nd ); hipFree ( Pd );
																																																											}//End of MatrixMul function	

																																																											//Start of getMatWidth => Get width i.e. # of columns
																																																											int getMatWidth( char *filename ){
																																																												int width;
																																																													//assumes space separate integer values e.g. -1 23 4 -56 6 77
																																																														//assumes first integer in file is row, 2nd integer is column
																																																															FILE *ptr = fopen( filename, "r" );
																																																																if ( ptr == 0 ){
																																																																		printf( "\n could not open file %s \n", filename );
																																																																				width = 0;
																																																																					}
																																																																						else{
																																																																								fscanf( ptr, "%d", &width  );
																																																																									}
																																																																										fclose( ptr );
																																																																											return width;
																																																																											}//end of getMatWidth function

																																																																											//Start of getMatHeight => Get height i.e. # of rows
																																																																											int getMatHeight( char *filename ){
																																																																												int height, dummy;
																																																																													//assumes space separate integer values e.g. -1 23 4 -56 6 77
																																																																														//assumes first integer in file is row, 2nd integer is column
																																																																															FILE *ptr = fopen( filename, "r" );
																																																																																if ( ptr == 0 ){
																																																																																		printf( "\n could not open file %s \n", filename );
																																																																																				height = 0;
																																																																																					}
																																																																																						else{
																																																																																								for ( int count = 1; count < 3; count++ ){
																																																																																											if ( count != 2 )
																																																																																															fscanf( ptr, "%d", &dummy );
																																																																																																		else
																																																																																																						fscanf( ptr, "%d", &dummy  );
																																																																																																										height = dummy;
																																																																																																												}
																																																																																																													}
																																																																																																														fclose( ptr );
																																																																																																															return height;
																																																																																																															}//end of getMatHeight function

																																																																																																															//START of loadMatrixFile function
																																																																																																															int  *loadMatrixFile( FILE *ptr, int cols, int rows ) {
																																																																																																																		int y = 1;
																																																																																																																					int x, *z;
																																																																																																																					//			int offset = 4;
																																																																																																																								//int w = 0;
																																																																																																																											fscanf( ptr, "%d", &x  );
																																																																																																																														while( !feof( ptr ) ){
																																																																																																																																		if ( y < 2 ){
																																																																																																																																							fscanf( ptr, "%d", &x );
																																																																																																																																												printf( "\n A: y: %d MatEl: %d \n", y, x );
																																																																																																																																																}
																																																																																																																																																				else {
																																																																																																																																																									fscanf( ptr, "%d", &z[ y - 2 ]  );
																																																																																																																																																														printf( " B: z[ w ]: %d \n", z[ y - 2 ] );
																																																																																																																																																																		}
																																																																																																																																																																						y++;
																																																																																																																																																																									}
																																																																																																																																																																										return z;
																																																																																																																																																																										}//END of loadMatrixFile function

																																																																																																																																																																										/*
																																																																																																																																																																										** END OF Auxiliary functions
																																																																																																																																																																										*/


																																																																																																																																																																										/*
																																																																																																																																																																										** START OF MAIN FUNCTION
																																																																																																																																																																										*/

																																																																																																																																																																										int main ( int argc, char *argv[ ] ) {
																																																																																																																																																																											
																																																																																																																																																																												char *filename1 = argv[ 1 ];
																																																																																																																																																																													char *filename2 = argv[ 2 ];
																																																																																																																																																																														int *matA; //holds first matrix
																																																																																																																																																																															int *matB; //holds sencond matrix
																																																																																																																																																																																int *matC;
																																																																																																																																																																																	int  sqWidth;

																																																																																																																																																																																		if ( argc != 3 ) /* argc should be 4 for correct execution */ {
																																																																																																																																																																																				/* We print argv[0] assuming it is the program name */
																																																																																																																																																																																						printf( "\nusage: %s matrixFile1 matrixFile2 \n\n", argv [0 ] );
																																																																																																																																																																																							}
																																																																																																																																																																																								else {

																																																																																																																																																																																										//returns # of cols of matrix, zero otherwise
																																																																																																																																																																																												int matWidthA = getMatWidth ( filename1  );
																																																																																																																																																																																														//get # of rows of matrix, zero otherwise
																																																																																																																																																																																																int matHeightA = getMatHeight( filename1 );

																																																																																																																																																																																																		//returns # of cols of matrix, zero otherwise
																																																																																																																																																																																																				int matWidthB = getMatWidth ( filename2  );
																																																																																																																																																																																																						//get # of rows of matrix, zero otherwise
																																																																																																																																																																																																								int matHeightB = getMatHeight( filename2 );
																																																																																																																																																																																																										
																																																																																																																																																																																																												//load matrices from files
																																																																																																																																																																																																														FILE *ptr1 = fopen( argv[ 1 ], "r" );
																																																																																																																																																																																																																FILE *ptr2 = fopen( argv[ 2 ], "r" );
																																																																																																																																																																																																																		if ( ptr1 == 0 && ptr2 == 0 )
																																																																																																																																																																																																																					printf( "\n could not open one of the following files: %s %s \n", argv[ 1 ], argv[ 2 ] );
																																																																																																																																																																																																																							else
																																																																																																																																																																																																																									{	
																																																																																																																																																																																																																												matA = loadMatrixFile( ptr1, matWidthA, matHeightA );
																																																																																																																																																																																																																															matB = loadMatrixFile( ptr2, matWidthB, matHeightB );
																																																																																																																																																																																																																																		fclose( ptr1 );
																																																																																																																																																																																																																																					fclose( ptr2 );
																																																																																																																																																																																																																																						      //Print matrix P
																																																																																																																																																																																																																																						      		        for ( int w = 0; w < matWidthA * matWidthA; w++ ){
																																																																																																																																																																																																																																												        printf( "\n" );
																																																																																																																																																																																																																																																	printf( " %d: %d  ", w, matA[ w ] );
																																																																																																																																																																																																																																																					printf( "\n" );
																																																																																																																																																																																																																																																								}
																																																																																																																																																																																																																																																										}
																																																																																																																																																																																																																																																												printMatrix( matA, matWidthA, matHeightA );
																																																																																																																																																																																																																																																														printMatrix( matB, matWidthB, matHeightB );

																																																																																																																																																																																																																																																																if ( matWidthB > matWidthA )
																																																																																																																																																																																																																																																																			sqWidth = matWidthB;
																																																																																																																																																																																																																																																																					else
																																																																																																																																																																																																																																																																								sqWidth = matWidthB;
																																																																																																																																																																																																																																																																										printf( "\n DEBUG \n" );
																																																																																																																																																																																																																																																																												//make matrices square ones first before multiplying
																																																																																																																																																																																																																																																																														//MatrixMul( matA, matB, matC, sqWidth );
																																																																																																																																																																																																																																																																																//printMatrix( matC, sqWidth, sqWidth );
																																																																																																																																																																																																																																																																																		fclose(ptr1); fclose( ptr2 );
																																																																																																																																																																																																																																																																																			}
																																																																																																																																																																																																																																																																																			}
																																																																																																																																																																																																																																																																																			/*
																																																																																																																																																																																																																																																																																			** END OF MAIN FUNCTION
																																																																																																																																																																																																																																																																																			*/
