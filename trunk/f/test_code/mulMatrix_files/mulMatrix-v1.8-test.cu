
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <cuda.h>
//#include "cuda_runtime_api.h"
//#include <stdint.h>
//#include <stdlib.h>

//This is the working matrix multiplication code - very basic
/*
Done:
- printing of matrix in a more pleasant manner using printMatrix function
- command line arguments
- opens matrix files and reads the matrix successfully
- working array passing from main to auxiliary (loadMatrixFile) function :)
- fixed printing of matrix
- fixed erroneous matrix values by moving loading into host matrix multiplication function!
- basic move towards SN P simulation: multiplication of s0 and Msnp

Problems:
- (fixed)  MatA and MatB values are overlapping and erroneous
*/


// START of Auxiliary functions

//START vector addition kernel function
__global__ void MatrixAddKernel ( int  *Md, int *Nd, int *Pd, int Width ){
        int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ k * Width + tx ];
		Pvalue += Mdelement + Ndelement;
	}
        Pd[ ty * Width + tx  ] = Pvalue;
}							
//END of kernel addition


//Start of kernel multiplication
__global__ void MatrixMulKernel ( int  *Md, int *Nd, int *Pd, int Width ){
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		int Mdelement = Md[ ty * Width + k ];
		int Ndelement = Nd[ k * Width + tx ];
		Pvalue += Mdelement * Ndelement;
	}

	Pd[ ty * Width + tx  ] = Pvalue;
}
//End of kernel multiplication


//function to print matrix
void printMatrix ( int *M, int rows, int columns ){
	//assumes matrix is in row-major format
	int index;
	printf ( "\n \n " );
	for ( int v = 0; v < rows; v++  ){
	//assumes a square matrix
		for ( int w = 0; w < columns; w++   ) {
			index = v * columns + w;
			printf ( " %02d", M[ index ]  );
		}
		printf ( " \n\n " );
	}
}//End of printMatrix function

//Start of getMatWidth => Get width i.e. # of columns
int getMatWidth( char *filename ){
	int width;
	//assumes space separate integer values e.g. -1 23 4 -56 6 77
	//assumes first integer in file is row, 2nd integer is column
	FILE *ptr = fopen( filename, "r" );
	if ( ptr == 0 ){
		printf( "\n could not open file %s \n", filename );
		width = 0;
	}
	else{
		fscanf( ptr, "%d", &width  );
	}
	fclose( ptr );
	return width;
}//end of getMatWidth function


//Start of getMatHeight => Get height i.e. # of rows
int getMatHeight( char *filename ){
	int height, dummy;
	//assumes space separate integer values e.g. -1 23 4 -56 6 77
	//assumes first integer in file is row, 2nd integer is column
	FILE *ptr = fopen( filename, "r" );
	if ( ptr == 0 ){
		printf( "\n could not open file %s \n", filename );
		height = 0;
	}
	else{
		for ( int count = 1; count < 3; count++ ){
			if ( count != 2 )
				fscanf( ptr, "%d", &dummy );
		else
			fscanf( ptr, "%d", &dummy  );
			height = dummy;
		}
	}
	fclose( ptr );
	return height;
}//end of getMatHeight function


//START of loadMatrixFile
void loadMatrixFile( char *filename, int *z, int matWidth, int matHeight ){
	int y = 0;
	int w = 0;
	int x;
	int offset = 0;
	FILE *ptr1 = fopen( filename, "r" );
//	int *z = ( int * )malloc( sizeof( ( matWidth * matHeight ) ) );
	//int z[ ( matWidth * matHeight ) + offset ] ;
	fscanf( ptr1, " %d", &x  );
	while( !feof( ptr1 ) && y < ( matWidth * matHeight ) + 1 ){
		if ( y > offset ){
			fscanf( ptr1, " %d", &z[ w - offset ]  );
			//printf( " B: z[ %d ]: %d \n", w, z[ w - offset ] );
			w++;
		}
		else{
			fscanf( ptr1, " %d", &x );
		}
		y++;
	}
	fclose( ptr1 );
//	x = y = w = 0;
//	array = &z[ 0 ];
//	free( z );
}
//END of loadMatrixFile


//Start of matrix multiplication host function MatrixMul
void MatrixMul( char *filename1, char *filename2, int Width /*, int *M, int *N, int  *P, int Width*/ ){
	int size = Width * Width * sizeof( int );
	int *Md, *Nd, *Pd;

	dim3 dimBlock( Width, Width );
	dim3 dimGrid( 1, 1 );

			//load matrices from files
			//get heigh/rows and width/columns of matrices
/*			int matWidthA = getMatWidth ( filename1  );
			int matHeightA = getMatHeight ( filename1  );
			
			int matWidthB = getMatWidth ( filename2  );
			int matHeightB = getMatHeight ( filename2  ); */
			
			int *matA = ( int * )malloc( size );
			//printf( "Width and height of Matrix A: %d %d and init values are\n", Width, Width );
			//printMatrix( matA, Width, Width );
			loadMatrixFile( filename1, matA, Width, Width );

			printf( " \nMatrix A after loading from file: \n" );
			printMatrix( matA, Width, Width );
			
			int *matB = ( int * )malloc( size );
			//printf( "Width and height of Matrix B: %d %d and init values are\n", Width, Width );
			//printMatrix( matB, Width, Width );
			loadMatrixFile( filename2, matB, Width, Width );
		
			printf( " \nMatrix B after loading from file: \n" );
			printMatrix( matB, Width, Width );
			
			//assumes a square matrix
			int *matC = ( int * )malloc( size );
			
/*			printf( "A: \n" );
			for ( int w = 0; w < Width * Width + 10; w++ ){
			        printf( "%d: %d \n",w,  matA[ w ] );
			}
			printf( "\n" );
*/
			//printf( " \nMatrix C initially: \n" );
			//printMatrix( matC, Width, Width );


	hipMalloc( (void**) &Md, size );
	hipMemcpy( Md, matA, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Nd, size );
        hipMemcpy( Nd, matB, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Pd, size );	
	
	MatrixMulKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );

	hipMemcpy( matC, Pd, size, hipMemcpyDeviceToHost );

			printf( " \nMatrix C finally: \n" );
			printMatrix( matC, Width, Width );

	free( matA ); free( matB ); free( matC );
	hipFree( Md ); hipFree( Nd ); hipFree ( Pd );
}
//End of Matrix multiplication function MatrixMul


//END of Auxiliary functions


//START of Main function
int main ( int argc, char *argv[ ] ) {
	int offset = 2;

	if ( argc != 5 ) {
		printf( "\nusage: %s configurationVector spikingVector spikingTransitionMatrix squarematrixwidth\n\n", argv [ 0 ] );
	}
	else {
		char *confVec = argv[ 1 ];
		char *spikVec = argv[ 2 ];
		char *spikTransMat = argv[ 3 ];
		int width = atoi( argv[ 4 ] );
		
		printf( "\nYou have entered files %s, %s, and %s and square matrix width %d \n", spikVec, confVec, spikTransMat, width );

		//load matrices from files
		FILE *ptr1 = fopen( confVec, "r" );
		FILE *ptr2 = fopen( spikVec, "r" );
		FILE *ptr3 = fopen( spikTransMat, "r" );

		if ( ptr1 == 0 && ptr2 == 0 && ptr3 == 0 )
			printf( "\n could not open one of the following files: %s %s \n", argv[ 1 ], argv[ 2 ] );
		else {
			MatrixMul( spikVec, spikTransMat, width );
		}
		fclose( ptr1 ); fclose( ptr2 ); fclose( ptr3 );
	}
}
//END of Main function
