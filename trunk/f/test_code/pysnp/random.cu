
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
 
int randomize( int maxIntVal, int seed1 ){
//	int x;
	time_t seconds;
	time( &seconds );
	seconds += seconds + seed1;
	srand( ( unsigned int ) seconds );
//	srand( ( unsigned ) time( NULL ) );
//	srand( ( unsigned ) seed );
//	int max = atoi( argv[ 1 ] );
	return rand( ) % maxIntVal + 1;	
}


int main( int argc, char *argv[ ] )
{
	if ( argc != 2 )
		printf( "\nUsage:\n %s maxrandomvalue\n", argv[ 0 ] );
	else {	
		int randMax = atoi( argv[ 1 ] );
		int x, randNum;
		for ( x = 0; x < 50; x++ ){
			randNum = randomize( randMax, x * randMax );
			printf( "%i ", randNum );
		}
	}
}
