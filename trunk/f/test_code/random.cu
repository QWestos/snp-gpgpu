/*
Steven Billington
January 17, 2003
exDice.cpp
Program rolls two dice with random
results.
*/
/*
Header: iostream
Reason: Input/Output stream
Header: stdlib
Reason: For functions rand and srand
Header: time.h
Reason: For function time, and for data type time_t
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
/*
These constants define our upper
and our lower bounds. The random numbers
will always be between 1 and 6, inclusive.
*/
const int LOW = 1;
const int HIGH = 100;
int main()
{
/*
Variables to hold random values
for the first and the second die on
each roll.
*/
int first_die, sec_die;
/*
Declare variable to hold seconds on clock.
*/
time_t seconds;
/*
Get value from system clock and
place in seconds variable.
*/
time(&seconds);
/*
Convert seconds to a unsigned
integer.
*/
srand((unsigned int) seconds);
/*
Get first and second random numbers.
*/
first_die = rand() % (HIGH - LOW + 1) + LOW;
sec_die = rand() % (HIGH - LOW + 1) + LOW;
/*
Output first roll results.
*/
printf( "Your roll is %d, %d \n", first_die, sec_die );
//cout<< "Your roll is (" << first_die << ", "
//<< sec_die << "}" << endl << endl;
/*
Get two new random values.
*/
first_die = rand() % (HIGH - LOW + 1) + LOW;
sec_die = rand() % (HIGH - LOW + 1) + LOW;
/*
Output second roll results.
*/
printf( "My roll is %d, %d \n", first_die, sec_die );
//cout<< "My roll is (" << first_die << ", "
//<< sec_die << "}" << endl << endl;
return 0;
}

/*#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
 
int randomize( int maxIntVal ){
//	int x;
	time_t seconds;
	time( &seconds );
	srand( ( unsigned int ) seconds );
//	int max = atoi( argv[ 1 ] );
	return rand( ) % maxIntVal + 1;	
}


int main( int argc, char *argv[ ] )
{
	if ( argc != 2 )
		printf( "\nUsage:\n %s maxrandomvalue\n", argv[ 0 ] );
	else {	
		int max = atoi( argv[ 1 ] );
		int x, randNum;
		for ( x = 0; x < 50; x++ ){
			randNum = randomize( max );
			printf( "%i ", randNum );
		}
	}
}
*/
/*
# random numbers in a range
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
 
void main()
{
int x ;
srand((unsigned)time(NULL));

for(x=0;x<=100;x++)
printf("%i\t",rand()%10 + 1);

}
*/
