/*
#include <iostream>
#include <cstdlib>
#include <time.h>

//upper and lower bounds, inclusive
const int LOW = 1;
const int HIGH = 6;
int main()
{
int first_die, sec_die;

//Declare variable to hold seconds on clock.

time_t seconds;

//Get value from system clock and place in seconds variable.

time(&seconds);

// Convert seconds to a unsigned int

srand((unsigned int) seconds);
first_die = rand() % (HIGH - LOW + 1) + LOW;
sec_die = rand() % (HIGH - LOW + 1) + LOW;
printf( "Your roll is %d, %d \n", first_die, sec_die );
return 0;
}
*/

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
 
int randomize( int maxIntVal, int seed1 ){
//	int x;
	time_t seconds;
	time( &seconds );
	seconds += seconds + seed1;
	srand( ( unsigned int ) seconds );
//	srand( ( unsigned ) time( NULL ) );
//	srand( ( unsigned ) seed );
//	int max = atoi( argv[ 1 ] );
	return rand( ) % maxIntVal + 1;	
}


int main( int argc, char *argv[ ] )
{
	if ( argc != 2 )
		printf( "\nUsage:\n %s maxrandomvalue\n", argv[ 0 ] );
	else {	
		int randMax = atoi( argv[ 1 ] );
		int x, randNum;
		for ( x = 0; x < 50; x++ ){
			randNum = randomize( randMax, x * randMax );
			printf( "%i ", randNum );
		}
	}
}

/*
// random numbers in a range
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
 
int main( void )
{
int x ;
srand((unsigned)time(NULL));

for(x=0;x<=100;x++)
printf("%i\t",rand()%99 + 1);

}
*/
