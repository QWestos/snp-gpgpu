#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <stdint.h>
#include <stdlib.h>

// DO NOT EDIT THIS!!!
//This is the working matrix multiplication code - very basic

//#define Width 4

__global__ void MatrixMulKernel ( float  *Md, float *Nd, float *Pd, int Width ){
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	float Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		float Mdelement = Md[ ty * Width + k ];
		float Ndelement = Nd[ k * Width + tx ];
		Pvalue += Mdelement * Ndelement;
	}

	Pd[ ty * Width + tx  ] = Pvalue;
}

void MatrixMul( float *M, float *N, float  *P, int Width ){
	int size = Width * Width * sizeof( float );
	float *Md, *Nd, *Pd;

	hipMalloc( (void**) &Md, size );
	hipMemcpy( Md, M, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Nd, size );
        hipMemcpy( Nd, N, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Pd, size );
	
	dim3 dimBlock( Width, Width );
	dim3 dimGrid( 1, 1 );
	
	MatrixMulKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );

	hipMemcpy( P, Pd, size, hipMemcpyDeviceToHost );

	for ( int w = 0; w < Width * Width; w++ ){
		printf( "\n" );
		printf( " %d: %f  ", w, P[w] );
		printf( "\n" );
	}

	hipFree( Md ); hipFree( Nd ); hipFree ( Pd );
}

int main ( void ) {
	int Width = 4;
	
	float A[ Width * Width ];

	for ( int x = 0; x < Width * Width; x++ ){
		A[ x ] = 2;
	}
	
	float B[ Width * Width ];
	for ( int z = 0; z < Width * Width; z++ ){
		B[ z ] = 2;
	}
	
	float C[ Width * Width ];

	MatrixMul( A, B, C, Width );
}

