#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <stdint.h>
#include <stdlib.h>

//#define Width 4

//Matrix multiplication kernel
__global__ void MatrixMulKernel ( float  *Md, float *Nd, float *Pd, int Width ){
	//2D thread ID
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//Pvalue stores Pd element computed by thread
	float Pvalue = 0;
	for ( int k = 0; k < Width; ++k ){
		float Mdelement = Md[ ty * Width + k ];
		float Ndelement = Nd[ k * Width + tx ];
		Pvalue += Mdelement * Ndelement;
	}

	//Write matrix to device memory; each thread writes one element
	Pd[ ty * Width + tx  ] = Pvalue;
}

void MatrixMul( float *M, float *N, float  *P, int Width ){
	int size = Width * Width * sizeof( float );
	float *Md, *Nd, *Pd;

	//Transfer M, N to device
	hipMalloc( (void**) &Md, size );
	hipMemcpy( Md, M, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Nd, size );
        hipMemcpy( Nd, N, size, hipMemcpyHostToDevice );
	hipMalloc( (void**) &Pd, size );
	
	//invoke kernel
	dim3 dimBlock( Width, Width );
	dim3 dimGrid( 1, 1 );
	
	//Launch kernel
	MatrixMulKernel<<< dimGrid, dimBlock >>>( Md, Nd, Pd, Width );

	//transfer from device to host
	hipMemcpy( P, Pd, size, hipMemcpyDeviceToHost );

	//Print matrix P
	for ( int w = 0; w < Width * Width; w++ ){
		printf( "\n" );
		printf( " %d: %f  ", w, P[w] );
		printf( "\n" );
	}

	//Free device matrices
	hipFree( Md ); hipFree( Nd ); hipFree ( Pd );
}

int main ( void ) {
	int Width = 4;
	float A[ Width * Width ];

	for ( int x = 0; x < Width * Width; x++ ){
		A[ x ] = 2;
	}
	
	float B[ Width * Width ];
	for ( int z = 0; z < Width * Width; z++ ){
		B[ z ] = 2;
	}
	
	float C[ Width * Width ];
	//= { 1, 1, 1, 1,1,1,1,1,1,1,1,1,1,1,1,1  };
	//float B[ Width * Width ] = { 1, 1, 1, 1,1,1,1,1,1,1,1,1,1,1,1,1  };
	//float C[ Width * Width ] = { 1, 1, 1, 1,1,1,1,1,1,1,1,1,1,1,1,1  };

	MatrixMul( A, B, C, Width );
}

void printMatrix ( float *A, float *B, float *C, int Width, int Height ){
	
}
